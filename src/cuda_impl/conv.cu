#include "hip/hip_runtime.h"
/*
 * @Author: liushijie
 * @Date: 2020-07-08 11:10:20
 * @LastEditTime: 2020-07-09 19:04:11
 * @LastEditors: liushijie
 * @Description:
 * @FilePath: /LightLR/src/cuda_impl/conv.cu
 */

#include "cuda/cuda_conv.h"
#include "macro.h"
#include <hip/hip_runtime.h>

namespace dl {
namespace cuda {
__global__ void CudaConv2DKernal(float *x, float *weight, float *res, int h,
                                 int w, int kernel_size, int c_in, int stride) {
    // dim3 grid_size(c_out, n);
    // dim3 block_size(w_out, h_out, c_in);

    int c_out = gridDim.x;
    int w_out = blockDim.x;
    int h_out = blockDim.y;

    int out_idx = Expand(threadIdx.x, w_out, threadIdx.y, h_out, blockIdx.x,
                         c_out, blockIdx.y);
    float sum = 0.0f;
    for (int k1 = 0; k1 < kernel_size; ++k1) {
        for (int k2 = 0; k2 < kernel_size; ++k2) {
            for (int c = 0; c < c_in; ++c) {
                sum += weight[Expand(k1, kernel_size, k2, kernel_size,
                                              c, c_in, blockIdx.x)] *
                                x[Expand(threadIdx.x * stride + k1, w,
                                         threadIdx.y * stride + k2, h, c, c_in,
                                         blockIdx.y)];
            }
        }
    }
    res[out_idx] = sum;
}

void CudaConv2D(const Tensor &x, const Tensor &weight, Tensor *out,
                Tensor *flatten_x, int stride, int padding,
                const std::string &padding_mode) {
    CHECK_EQ(x.shape().size(), 4);
    CHECK_EQ(weight.shape().size(), 4);
    LOG_DEBUG("\n\tTensor CudaConv2D args:\n\t\tx shape:(%d, %d, %d, "
              "%d)\n\t\tweight shape:(%d, %d, %d, "
              "%d)\n\t\tstride:%d\n\t\tpadding:%d",
              x.shape()[0], x.shape()[1], x.shape()[2], x.shape()[3],
              weight.shape()[0], weight.shape()[1], weight.shape()[2],
              weight.shape()[3], stride, padding);

    CHECK_EQ(x.shape()[1], weight.shape()[1]);

    int c_out = weight.shape()[0];
    int c_in  = weight.shape()[1];
    int k     = weight.shape()[2];
    int n     = x.shape()[0];
    int h     = x.shape()[2];
    int w     = x.shape()[3];
    int h_out = (h + 2 * padding - k) / stride + 1;
    int w_out = (w + 2 * padding - k) / stride + 1;
    LOG_DEBUG("(h_out: %d, w_out: %d)", h_out, w_out);
    CHECK_EQ(out->shape()[0], n);
    CHECK_EQ(out->shape()[1], c_out);
    CHECK_EQ(out->shape()[2], h_out);
    CHECK_EQ(out->shape()[3], w_out);

    float *dev_x, *dev_weight, *dev_res;
    gpuErrchk(hipMalloc(&dev_x, sizeof(float) * x.size()));
    gpuErrchk(hipMalloc(&dev_weight, sizeof(float) * weight.size()));
    gpuErrchk(hipMalloc(&dev_res, sizeof(float) * n * c_out * h_out * w_out));
    gpuErrchk(hipMemcpy(dev_x, x.data(), sizeof(float) * x.size(),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_weight, weight.data(),
                         sizeof(float) * weight.size(),
                         hipMemcpyHostToDevice));

    dim3 grid_size(c_out, n);
    dim3 block_size(w_out, h_out);
    CudaConv2DKernal<<<grid_size, block_size>>>(dev_x, dev_weight, dev_res, h,
                                                w, k, c_in, stride);
    gpuErrchk(hipMemcpy(out->data(), dev_res, sizeof(float) * out->size(),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(dev_x));
    gpuErrchk(hipFree(dev_weight));
    gpuErrchk(hipFree(dev_res));
}
}
}