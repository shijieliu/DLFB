#include "hip/hip_runtime.h"
/*
 * @Author: liushijie
 * @Date: 2020-07-06 18:02:03
 * @LastEditTime: 2020-07-09 20:39:46
 * @LastEditors: liushijie
 * @Description:
 * @FilePath: /LightLR/src/cuda_impl/mul.cu
 */
#include "cuda/cuda_mul.h"
#include "macro.h"
#include <hip/hip_runtime.h>

namespace dl {
namespace cuda {
__global__ void CudaMulKernal(float *x, float *y, float *res, int len) {
    int begin_idx   = blockIdx.x * blockDim.x + threadIdx.x;
    int read_offset = blockDim.x * gridDim.x;
    for (int i = begin_idx; i < len; i += read_offset) {
        res[i] = x[i] * y[i];
    }
}

void CudaMul(const Tensor &x, const Tensor &y, Tensor *res) {
    CHECK_EQ(x.size(), res->size());
    CHECK_EQ(y.size(), res->size());

    int    len = x.size();
    float *devx, *devy, *dev_res;
    gpuErrchk(hipMalloc(&devx, sizeof(float) * len));
    gpuErrchk(hipMalloc(&devy, sizeof(float) * len));
    gpuErrchk(hipMalloc(&dev_res, sizeof(float) * len));

    gpuErrchk(hipMemcpy(devx, x.data(), sizeof(float) * len,
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devy, y.data(), sizeof(float) * len,
                         hipMemcpyHostToDevice));
    CudaMulKernal<<<32, 1024>>>(devx, devy, dev_res, len);
    gpuErrchk(hipMemcpy(res->data(), dev_res, sizeof(float) * len,
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(devx));
    gpuErrchk(hipFree(devy));
    gpuErrchk(hipFree(dev_res));
}

template <size_t TileSize>
__global__ void CudaMatKernal(float *x, float *y, float *res, int row, int col,
                              int len) {
    __shared__ float shared_x[TileSize][TileSize];
    __shared__ float shared_y[TileSize][TileSize];

    int   tx    = threadIdx.x;
    int   ty    = threadIdx.y;
    int   bx    = blockIdx.x;
    int   by    = blockIdx.y;
    int   r     = Expand(ty, TileSize, by);
    int   c     = Expand(tx, TileSize, bx);
    float value = 0.;
    for (int m = 0; m < (len - 1) / TileSize + 1; ++m) {
        if (r < row && Expand(tx, TileSize, m) < len) {
            shared_x[tx][ty] =
                x[Expand(Expand(tx, TileSize, m), len, r)]; // x shape(row, len)
                                                            // x idx (r,
                                                            // Expand(tx,
                                                            // TileSize, m))
        } else {
            shared_x[tx][ty] = 0;
        }
        if (c < col && Expand(ty, TileSize, m) < len) {
            shared_y[tx][ty] =
                y[Expand(c, col, Expand(tx, TileSize, m))]; // y shape (len,
                                                            // col) y idx
                                                            // (Expand(tx,
                                                            // TileSize, m) , c)
        } else {
            shared_y[tx][ty] = 0;
        }

        __syncthreads();
        for (int k = 0; k < TileSize; ++k) {
            value += shared_x[tx][k] * shared_y[k][ty];
        }
        __syncthreads();
    }
    res[Expand(c, col, r)] = value;
}

void CudaMat(const Tensor &lhs, const Tensor &rhs, Tensor *res) {
    CHECK_EQ(lhs.shape().size(), 2);
    CHECK_EQ(lhs.shape()[1], rhs.shape()[0]);

    int row = lhs.shape()[0];
    int col = rhs.shape()[1];
    int len = lhs.shape()[1];
    CHECK_EQ(res->size(), row * col);

    float *devx, *devy, *dev_res;
    gpuErrchk(hipMalloc(&devx, sizeof(float) * lhs.size()));
    gpuErrchk(hipMalloc(&devy, sizeof(float) * rhs.size()));
    gpuErrchk(hipMalloc(&dev_res, sizeof(float) * res->size()));

    gpuErrchk(hipMemcpy(devx, lhs.data(), sizeof(float) * lhs.size(),
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devy, rhs.data(), sizeof(float) * rhs.size(),
                         hipMemcpyHostToDevice));

    constexpr int tile_size = 16;
    dim3 grid_size((row - 1) / tile_size + 1, (col - 1) / tile_size + 1);
    dim3 block_size(tile_size, tile_size);
    CudaMatKernal<tile_size><<<grid_size, block_size>>>(devx, devy, dev_res, row, col, len);
    gpuErrchk(hipMemcpy(res->data(), dev_res, sizeof(float) * res->size(),
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(devx));
    gpuErrchk(hipFree(devy));
    gpuErrchk(hipFree(dev_res));
}
}
}