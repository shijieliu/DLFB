#include "hip/hip_runtime.h"
/*
 * @Author: liushijie
 * @Date: 2020-07-06 18:02:03
 * @LastEditTime: 2020-07-08 11:50:41
 * @LastEditors: liushijie
 * @Description:
 * @FilePath: /LightLR/src/cuda_impl/add.cu
 */
#include "cuda/cuda_add.h"
#include "macro.h"
#include <hip/hip_runtime.h>

namespace dl {
namespace cuda {
__global__ void CudaAddKernal(float *x, float *y, float *res, int len) {
    int begin_idx   = blockIdx.x * blockDim.x + threadIdx.x;
    int read_offset = blockDim.x * gridDim.x;
    for (int i = begin_idx; i < len; i += read_offset) {
        res[i] = x[i] + y[i];
    }
}

void CudaAdd(const Tensor &x, const Tensor &y, Tensor *res) {
    CHECK_EQ(x.size(), res->size());
    CHECK_EQ(y.size(), res->size());

    int    len = x.size();
    float *devx, *devy, *dev_res;
    gpuErrchk(hipMalloc((void **) &devx, sizeof(float) * len));
    gpuErrchk(hipMalloc((void **) &devy, sizeof(float) * len));
    gpuErrchk(hipMalloc((void **) &dev_res, sizeof(float) * len));

    gpuErrchk(hipMemcpy(devx, x.data(), sizeof(float) * len,
                         hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devy, y.data(), sizeof(float) * len,
                         hipMemcpyHostToDevice));
    CudaAddKernal<<<32, 1024>>>(devx, devy, dev_res, len);
    gpuErrchk(hipMemcpy(res->data(), dev_res, sizeof(float) * len,
                         hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(devx));
    gpuErrchk(hipFree(devy));
    gpuErrchk(hipFree(dev_res));
}
}
}