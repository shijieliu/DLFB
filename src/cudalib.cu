#include "hip/hip_runtime.h"
/*
 * @Author: liushijie
 * @Date: 2020-07-06 18:02:03
 * @LastEditTime: 2020-07-06 18:48:50
 * @LastEditors: liushijie
 * @Description: 
 * @FilePath: /LightLR/src/cudalib.cu
 */ 
#include <hip/hip_runtime.h>
#include "cuda/cudalib.h"
#include "macro.h"

namespace dl {
namespace cuda {
__global__ void CudaAddKernal(float *x, float *y, float *res, int len) {
    int begin_idx   = blockIdx.x * blockDim.x + threadIdx.x;
    int read_offset = blockDim.x * gridDim.x;
    for (int i = begin_idx; i < len; i += read_offset) {
        res[i] = x[i] + y[i];
    }
}

void CudaAdd(const float *x, const float *y, float *res, int len) {
    float *devx, *devy, *dev_res;
    gpuErrchk(hipMalloc((void **)&devx, sizeof(float) * len));
    gpuErrchk(hipMalloc((void **)&devy, sizeof(float) * len));
    gpuErrchk(hipMalloc((void **)&dev_res, sizeof(float) * len));

    gpuErrchk(hipMemcpy(devx, x, sizeof(float) * len, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devy, y, sizeof(float) * len, hipMemcpyHostToDevice));
    CudaAddKernal<<<BLOCK_SIZE, GRID_SIZE>>>(devx, devy, dev_res, len);
    gpuErrchk(hipMemcpy(res, dev_res, sizeof(float) * len, hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(devx));
    gpuErrchk(hipFree(devy));
    gpuErrchk(hipFree(dev_res));
}

// __global__ void conv_cuda(double *out, const double *filter, int K, int C, int FW, int FH,
//                           const double *paddedImage, int W, int H) {

//     int k = threadIdx.x + (blockDim.x * blockIdx.x);
//     int x = threadIdx.y + (blockDim.y * blockIdx.y);
//     int y = threadIdx.z + (blockDim.z * blockIdx.z);

//     auto outIndex = Expand(y, H, x, W, k);

//     double sum = 0.0;

// #pragma unroll 4
//     for (int c = 0; c < C; ++c) {
//         for (int j = 0; j < FH; ++j) {
//             for (int i = 0; i < FW; ++i) {
//                 auto filterIndex = Expand(FH - 1 - j, FH, FW - 1 - i, FW, c, C, k);
//                 auto imageIndex  = Expand(y + j, W + 2, x + i, H + 2, c);
//                 sum += (filter[filterIndex] * paddedImage[imageIndex]);
//             }
//         }
//     }
//     out[outIndex] = sum;
// }

// __global__ void conv_cuda_tiled(double *out, const double *filter, int K, int C, int FW, int FH,
//                                 const double *paddedImage, int W, int H) {

//     int k = threadIdx.x + (blockDim.x * blockIdx.x);
//     int x = threadIdx.y + (blockDim.y * blockIdx.y);
//     int y = threadIdx.z + (blockDim.z * blockIdx.z);

//     int tidx = threadIdx.y;
//     int tidy = threadIdx.z;

//     int X_BOUND = X_BLOCK + FW - 1;
//     int Y_BOUND = Y_BLOCK + FH - 1;

//     extern __shared__ double tile[];

//     for (int c = 0; c < C; ++c) {
//         tile[Expand(tidy, Y_BOUND, tidx, X_BOUND, c)] = paddedImage[Expand(y, W + 2, x, H + 2, c)];
//         // corner loads
//         if (tidx == X_BLOCK - 1 && tidy == Y_BLOCK - 1) {
//             for (int xx = 0; xx < FW; ++xx) {
//                 for (int yy = 0; yy < FH; ++yy) {
//                     if (xx == 0 and yy == 0)
//                         continue;
//                     tile[Expand(tidy + yy, Y_BOUND, tidx + xx, X_BOUND, c)] =
//                         paddedImage[Expand(y + yy, W + 2, x + xx, H + 2, c)];
//                 }
//             }
//         } // edge loads
//         else if (tidx == X_BLOCK - 1) {
//             for (int xx = 1; xx < FW; ++xx) {
//                 tile[Expand(tidy, Y_BOUND, tidx + xx, X_BOUND, c)] =
//                     paddedImage[Expand(y, W + 2, x + xx, H + 2, c)];
//             }
//         } else if (tidy == Y_BLOCK - 1) {
//             for (int yy = 1; yy < FH; ++yy) {
//                 tile[Expand(tidy + yy, Y_BOUND, tidx, X_BOUND, c)] =
//                     paddedImage[Expand(y + yy, W + 2, x, H + 2, c)];
//             }
//         }
//     }
//     __syncthreads();

//     auto   outIndex = Expand(y, H, x, W, k);
//     double sum      = 0.0;
// #pragma unroll 4
//     for (int c = 0; c < C; ++c) {
//         for (int j = 0; j < FH; ++j) {
//             for (int i = 0; i < FW; ++i) {
//                 auto filterIndex = Expand(FH - 1 - j, FH, FW - 1 - i, FW, c, C, k);
//                 auto imageIndex  = Expand(tidy + j, Y_BOUND, tidx + i, X_BOUND, c);
//                 sum += (filter[filterIndex] * tile[imageIndex]);
//             }
//         }
//     }
//     out[outIndex] = sum;
// }
}
}